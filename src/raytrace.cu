#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <glm/glm.hpp>
#include "common.h"
#include "Object.inl"
#include "Ray.inl"
#include "Utils.inl"

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
  r = glm::clamp(r, 0.0f, 1.0f);
  g = glm::clamp(g, 0.0f, 1.0f);
  b = glm::clamp(b, 0.0f, 1.0f);

  // notice switch red and blue to counter the GL_BGRA
  return (int(r*255.0)<<16) | (int(g*255.0)<<8) | int(b*255.0);
}
__device__ int rgbToInt(glm::vec3 c)
{
  c = glm::clamp(c, 0.0f, 1.0f);

  // notice switch red and blue to counter the GL_BGRA
  return (int(c.r*255.0)<<16) | (int(c.g*255.0)<<8) | int(c.b*255.0);
}

__global__ void raytraceKernel(
  const uint w, const uint h,
  const glm::vec3 campos, const glm::vec3 A, const glm::vec3 B, const glm::vec3 C,
  uint* pbo_out, 
  const Object::Object* scene, const uint sceneSize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  // calc camera rays
  glm::vec2 uv((float)x/w, (float)y/h);  
  Ray::Ray ray;
  ray.m_pos = campos+C + (2.0f*uv.x-1.0f)*A + (2.0f*uv.y-1.0f)*B;
  ray.m_dir = glm::normalize(ray.m_pos-campos);

  glm::vec3 lightDir(0.267261, 0.801784, 0.534522);
  Ray::Hit hit = Ray::intersectScene(ray, scene, sceneSize);
  //Ray::Hit hit = Ray::intersect(ray, *scene[0].m_mesh);

  glm::vec3 col;
  if (hit.m_id < 0) {
    col = ray.m_dir;
  }
  else {
    if (scene[hit.m_id].m_material.m_emit > 0.0f)
      col = scene[hit.m_id].m_material.m_color;
    else {
      col = scene[hit.m_id].m_material.m_color * scene[hit.m_id].m_material.m_brdf;
      col *= glm::max(glm::dot(lightDir,hit.m_nor),0.0f);
    }
  }

  pbo_out[idx] = rgbToInt(col);
}

__global__ void initBuffersKernel(
  const uint w, const uint h,
  const glm::vec3 campos, const glm::vec3 A, const glm::vec3 B, const glm::vec3 C,
  const float lensRadius, const float focalDist,
  glm::vec3* rand, uint* flags,
  Ray::Ray* rays, glm::vec3* col, 
  glm::vec3* film, uint filmIters)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;
  
  // calc camera rays
  glm::vec2 uv(
    (float)x/w + (2.0f*rand[idx].x-1.0f)/w, 
    (float)y/h + (2.0f*rand[idx].y-1.0f)/h
  );
  rays[idx].m_pos = campos+C + (2.0f*uv.x-1.0f)*A + (2.0f*uv.y-1.0f)*B;
  rays[idx].m_dir = glm::normalize(rays[idx].m_pos-campos);

  // focal blur
#ifdef FOCAL_BLUR
  glm::vec3 fpt = focalDist*rays[idx].m_dir+rays[idx].m_pos;
  glm::vec2 randdisk = lensRadius*Utils::randPointDisk(rand[idx].x,rand[idx].y,rand[idx].z);
  rays[idx].m_pos += randdisk.x*glm::normalize(A) + randdisk.y*glm::normalize(B);
  rays[idx].m_dir = glm::normalize(fpt-rays[idx].m_pos);
#endif

  // reset buffers
  col[idx] = glm::vec3(1.0f);

  if (filmIters==1)
    film[idx] = glm::vec3(0.0f);

  flags[idx] = THFL_NONE | THFL_PATH_RUN;
}

__global__ void calcColorKernel(
  const uint w, const uint h, const float time,
  const Object::Object* scene, const uint sceneSize,
  glm::vec3* rand,
  uint* flags,
  Ray::Ray* rays,
  glm::vec3* col,
  const int depth)
{

  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  if (!flags[idx]&THFL_PATH_RUN)
    return;

  // intersection test
  Ray::Hit hit = Ray::intersectScene(rays[idx], scene, sceneSize);
  
  // intersects nothing, kill path
  if( hit.m_id < 0 ) {
    col[idx] = glm::vec3(0.0f, 0.0f, 0.0f);   // BLACK
    flags[idx] &= !THFL_PATH_RUN;
    return;
  }

  // intersects light, kill path
  if (scene[hit.m_id].m_material.m_emit > 0.0f) {
    col[idx] *= scene[hit.m_id].m_material.m_color*scene[hit.m_id].m_material.m_emit;
    flags[idx] &= !THFL_PATH_RUN;

    //if (depth >= 2) {
    //  col[idx] = glm::vec3(0.0f, 1.0f, 0.0f);
    //}

    return;
  }

  // at max depth, not seen light, does not contribute color
  if (depth == PATH_DEPTH-1) {
    col[idx] = glm::vec3(0.0f, 0.0f, 0.0f);   // BLACK
    return;
  }

  // else, compute color, bounce

  col[idx] *= scene[hit.m_id].m_material.m_color;

  uint randidx = (idx + depth) % (w*h);
  rays[idx].m_dir = Material::bounce(scene[hit.m_id].m_material,
    rays[idx].m_dir, hit.m_nor, rand[randidx]);
  rays[idx].m_pos = hit.m_pos + EPS*rays[idx].m_dir;

}

__global__ void accumColorKernel(
  const uint w, const uint h,
  uint* pbo_out,
  glm::vec3* col,
  glm::vec3* film, const float filmIters)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  film[idx] += col[idx];

#ifdef GAMMA_CORRECT
  pbo_out[idx] = rgbToInt( glm::pow(film[idx]/filmIters, glm::vec3(1.0f/2.2f)) );
#else
  pbo_out[idx] = rgbToInt(film[idx]/filmIters);
#endif
  //pbo_out[idx] = rgbToInt(col[idx]);
}

__global__ void testRand(
  const uint w, const uint h,
  uint* pbo_out,
  glm::vec3* rand
  )
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;
  pbo_out[idx] = rgbToInt(rand[idx]);
}

extern "C"
void pathtrace(
  uint* pbo_out, const uint w, const uint h, const float time,
  const glm::vec3& campos, const glm::vec3& A, const glm::vec3& B, const glm::vec3& C,
  const float lensRadius, const float focalDist,
  const Object::Object* scene_d, const uint sceneSize,
  glm::vec3* rand_d,
  uint* flags_d,
  Ray::Ray* rays_d,
  glm::vec3* col_d,
  glm::vec3* film_d, const uint filmIters)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time*100.0f);
  hiprandGenerateUniform(gen, (float*)rand_d, 3*w*h);

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(w/block.x, h/block.y);

  initBuffersKernel<<<grid, block>>>(
    w,h,campos,A,B,C,lensRadius,focalDist,rand_d,flags_d,rays_d,col_d,film_d,filmIters
  );
  for (int i=0; i<PATH_DEPTH; ++i)
    calcColorKernel<<<grid, block>>>(
      w,h,time,scene_d,sceneSize,rand_d,flags_d,rays_d,col_d,i
    );
  accumColorKernel<<<grid, block>>>(w,h,pbo_out,col_d,film_d,filmIters);

  //testRand<<<grid, block>>>(w,h,pbo_out,rand_d);
}

extern "C"
void raytrace1(
  uint* pbo_out, const uint w, const uint h, const float time,
  const glm::vec3& campos, const glm::vec3& A, const glm::vec3& B, const glm::vec3& C,
  const Object::Object* scene_d, const uint sceneSize)
{
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(w/block.x,h/block.y);
  raytraceKernel<<<grid, block>>>(w,h,campos,A,B,C,pbo_out,scene_d,sceneSize);
}