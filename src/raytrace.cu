#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <glm/glm.hpp>
#include "common.h"
#include "Object.inl"
#include "Ray.inl"
#include "Utils.inl"

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
  r = glm::clamp(r, 0.0f, 1.0f);
  g = glm::clamp(g, 0.0f, 1.0f);
  b = glm::clamp(b, 0.0f, 1.0f);

  // notice switch red and blue to counter the GL_BGRA
  return (int(r*255.0)<<16) | (int(g*255.0)<<8) | int(b*255.0);
}
__device__ int rgbToInt(glm::vec3 c)
{
  c = glm::clamp(c, 0.0f, 1.0f);

  // notice switch red and blue to counter the GL_BGRA
  return (int(c.r*255.0)<<16) | (int(c.g*255.0)<<8) | int(c.b*255.0);
}

__global__ void testKernel(
  const uint w, const uint h,
  const glm::vec3 campos, const glm::vec3 A, const glm::vec3 B, const glm::vec3 C,
  uint* pbo_out, 
  const Object::Object* scene, const uint sceneSize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  // calc camera rays
  glm::vec2 uv((float)x/w, (float)y/h);  
  Ray::Ray ray;
  ray.m_pos = campos+C + (2.0f*uv.x-1.0f)*A + (2.0f*uv.y-1.0f)*B;
  ray.m_dir = glm::normalize(ray.m_pos-campos);

  glm::vec3 lightDir(0.267261, 0.801784, 0.534522);
  Ray::Hit hit = Ray::intersectScene(ray, scene, sceneSize);
  //Ray::Hit hit = Ray::intersect(ray, *scene[0].m_mesh);

  glm::vec3 col;
  if (hit.m_id < 0) {
    col = ray.m_dir;
  }
  else {
    col = scene[hit.m_id].m_material.m_color * scene[hit.m_id].m_material.m_brdf;
    col *= glm::max(glm::dot(lightDir,hit.m_nor),0.0f);
  }

  pbo_out[idx] = rgbToInt(col);
}

__global__ void initBuffersKernel(
  const uint w, const uint h,
  const glm::vec3 campos, const glm::vec3 A, const glm::vec3 B, const glm::vec3 C,
  const float lensRadius, const float focalDist,
  glm::vec3* rand, uint* flags,
  Ray::Ray* rays, glm::vec3* col, 
  glm::vec3* film, uint filmIters)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;
  
  // calc camera rays
  glm::vec2 uv(
    (float)x/w + (2.0f*rand[idx].x-1.0f)/w, 
    (float)y/h + (2.0f*rand[idx].y-1.0f)/h
  );
  rays[idx].m_pos = campos+C + (2.0f*uv.x-1.0f)*A + (2.0f*uv.y-1.0f)*B;
  rays[idx].m_dir = glm::normalize(rays[idx].m_pos-campos);

  // focal blur
#ifdef FOCAL_BLUR
  glm::vec3 fpt = focalDist*rays[idx].m_dir+rays[idx].m_pos;
  rays[idx].m_pos += lensRadius*rand[idx];
  rays[idx].m_dir = glm::normalize(fpt-rays[idx].m_pos);
#endif

  // reset buffers
  col[idx] = glm::vec3(1.0f);

  if (filmIters==1)
    film[idx] = glm::vec3(0.0f);

  flags[idx] = THFL_NONE | THFL_PATH_RUN;
}

__global__ void calcColorKernel(
  const uint w, const uint h, const float time,
  const Object::Object* scene, const uint sceneSize,
  glm::vec3* rand,
  uint* flags,
  Ray::Ray* rays,
  glm::vec3* col,
  const uint depth)
{

  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  if (!flags[idx]&THFL_PATH_RUN)
    return;

  // intersection test
  Ray::Hit hit = Ray::intersectScene(rays[idx], scene, sceneSize);
  
  // intersects nothing, kill path
  if( hit.m_id < 0 ) {
    col[idx] = glm::vec3(0.0f);
    flags[idx] &= !THFL_PATH_RUN;
    return;
  }
  
  // intersects light, kill path
  if (scene[hit.m_id].m_material.m_emit > 0.0f) {
    col[idx] *= scene[hit.m_id].m_material.m_color*scene[hit.m_id].m_material.m_emit;
    flags[idx] &= !THFL_PATH_RUN;
  }
  else {    
    // at max depth, no light, does not contribute color
    if (depth == PATH_DEPTH-1) {
      col[idx] = glm::vec3(0.0f);
      return;
    }

    col[idx] *= scene[hit.m_id].m_material.m_color;// * scene[hit.m_id].m_material.m_brdf;

    if (scene[hit.m_id].m_material.m_type == Material::DIFF) {
      rays[idx].m_dir = Utils::randVectorHem(rand[idx].x,rand[idx].y,hit.m_nor);
    }
    else if (scene[hit.m_id].m_material.m_type == Material::MIRR) {
      rays[idx].m_dir = glm::reflect(rays[idx].m_dir, hit.m_nor);
    }
    else if (scene[hit.m_id].m_material.m_type == Material::TRANS) {
      float n1 = 1.0f;
      float n2 = scene[hit.m_id].m_material.m_n;
      glm::vec3 nor = hit.m_nor;
      // if coming from inside
      if (glm::dot(rays[idx].m_dir,hit.m_nor) > 0.0f) {
        float temp = n1;
        n1 = n2;
        n2 = temp;
        nor = -nor;
      }
        
      float reflectance = Material::reflectance(hit.m_nor, rays[idx].m_dir, n1, n2);
      if (rand[idx].x < reflectance)
        rays[idx].m_dir = glm::reflect(rays[idx].m_dir, hit.m_nor);
      else
        rays[idx].m_dir = glm::refract(rays[idx].m_dir, nor, n1/n2);
    }
    rays[idx].m_pos = hit.m_pos + EPS*rays[idx].m_dir;
  }
}

__global__ void accumColorKernel(
  const uint w, const uint h,
  uint* pbo_out,
  glm::vec3* col,
  glm::vec3* film, const float filmIters)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  uint idx = y*w + x;

  film[idx] += col[idx];
  pbo_out[idx] = rgbToInt(film[idx]/filmIters);
  //pbo_out[idx] = rgbToInt(col[idx]);
}

extern "C"
void pathtrace(
  uint* pbo_out, const uint w, const uint h, const float time,
  const glm::vec3& campos, const glm::vec3& A, const glm::vec3& B, const glm::vec3& C,
  const float lensRadius, const float focalDist,
  const Object::Object* scene_d, const uint sceneSize,
  glm::vec3* rand_d,
  uint* flags_d,
  Ray::Ray* rays_d,
  glm::vec3* col_d,
  glm::vec3* film_d, const uint filmIters)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time*100.0f);
  hiprandGenerateUniform(gen, (float*)rand_d, 3*w*h);

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(w/block.x, h/block.y);

  initBuffersKernel<<<grid, block>>>(
    w,h,campos,A,B,C,lensRadius,focalDist,rand_d,flags_d,rays_d,col_d,film_d,filmIters
  );
  for (int i=0; i<PATH_DEPTH; ++i)
    calcColorKernel<<<grid, block>>>(
      w,h,time,scene_d,sceneSize,rand_d,flags_d,rays_d,col_d,i
    );
  accumColorKernel<<<grid, block>>>(w,h,pbo_out,col_d,film_d,filmIters);
}

extern "C"
void raytrace1(
  uint* pbo_out, const uint w, const uint h, const float time,
  const glm::vec3& campos, const glm::vec3& A, const glm::vec3& B, const glm::vec3& C,
  const Object::Object* scene_d, const uint sceneSize)
{
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(w/block.x,h/block.y);
  testKernel<<<grid, block>>>(w,h,campos,A,B,C,pbo_out,scene_d,sceneSize);
}